#include "cudaUtils.cuh"

template<typename PRECISION>
void FLB::copyDataFromDevice(size_t numPointsMesh, unsigned int numDimensions, unsigned int numVelocities, PRECISION* d_v, PRECISION* h_v)
{
  size_t fieldSize = numVelocities * numPointsMesh * sizeof(PRECISION);
  checkCudaErrors(hipMemcpy(h_v, d_v, fieldSize, hipMemcpyDeviceToHost));
}

void FLB::getInfoDevice(int& devID, hipDeviceProp_t& props)
{
  //int devID;
  //hipDeviceProp_t props;

  // Get GPU information
  hipGetDevice(&devID);
  hipGetDeviceProperties(&props, devID);

}
