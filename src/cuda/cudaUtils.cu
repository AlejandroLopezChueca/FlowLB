#include "hip/hip_runtime.h"
#include "cudaUtils.cuh"
#include "cudaInitData.cuh"
#include <cstddef>
#include <cstdint>
#include <cstdlib>
//#include "cudaUtils.h"

void FLB::CudaUtils::printInfoDevice(Fl_Simple_Terminal* terminal)
{
  int deviceCount = 0;
  int deviceID;
  int major = 0;
  int minor = 0;
  hipDeviceProp_t deviceProps;

  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0)
  {
    terminal -> printf("[GPU ERROR] No devices supporting CUDA detected\n");
    return;
  }
  else if (deviceCount == 1)
  {
    terminal -> printf("[GPU INFO] One device that supports CUDA has been detected\n");
  }
  else
  {
    terminal -> printf("[GPU INFO] More than one device that support CUDA has been detected\n");
  }
  
  checkCudaErrors(hipGetDevice(&deviceID));

  checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, deviceID));
  checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, deviceID));

  checkCudaErrors(hipGetDeviceProperties(&deviceProps, deviceID));

  terminal -> printf("[GPU INFO] CUDA device [%s] has %d Multi-Processors with compute capability %d.%d\n", deviceProps.name, deviceProps.multiProcessorCount, major, minor);

  // memory 
  size_t freeBytes, totalBytes;
  checkCudaErrors(hipMemGetInfo(&freeBytes, &totalBytes));
  size_t usedBytes = totalBytes - freeBytes;
  terminal -> printf("[GPU INFO] Total Memory (MB): %.2f\n", totalBytes / 1024.0 / 1024.0);
  terminal -> printf("[GPU INFO] Free Memory (MB): %.2f\n", freeBytes / 1024.0 / 1024.0);
  terminal -> printf("[GPU INFO] Used Memory (MB): %.2f\n", usedBytes / 1024.0 / 1024.0);

  uint32_t maxThreadsPerBlock = getMaxThreadsPerBlock();
  terminal -> printf("[GPU INFO] The maximum number of threads per block is %d\n", maxThreadsPerBlock);
}

uint32_t FLB::CudaUtils::getMaxThreadsPerBlock()
{
  int deviceID;
  checkCudaErrors(hipGetDevice(&deviceID));

  // first get the maximum number of threads per block
  int maxThreadsPerBlock;
  checkCudaErrors(hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, deviceID));
  return maxThreadsPerBlock;

}

dim3 FLB::CudaUtils::getBlockSize(int numDimensions)
{
  int deviceID;
  checkCudaErrors(hipGetDevice(&deviceID));

  // first it is necesary to know the maximum dmmesion of a block
  int maxBlockDimX, maxBlockDimY, maxBlockDimZ;
  checkCudaErrors(hipDeviceGetAttribute(&maxBlockDimX, hipDeviceAttributeMaxBlockDimX, deviceID));
  checkCudaErrors(hipDeviceGetAttribute(&maxBlockDimY, hipDeviceAttributeMaxBlockDimY, deviceID));
  checkCudaErrors(hipDeviceGetAttribute(&maxBlockDimZ, hipDeviceAttributeMaxBlockDimZ, deviceID));


}
dim3 FLB::CudaUtils::getGridSize(const int numDimensions, const FLB::Mesh* mesh, const dim3& blockSize)
{
  int deviceID;
  checkCudaErrors(hipGetDevice(&deviceID));

  // first it is necesary to know the maximum dmmesion of the grid
  int maxGridDimX, maxGridDimY, maxGridDimZ;
  checkCudaErrors(hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX, deviceID));
  checkCudaErrors(hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY, deviceID));
  checkCudaErrors(hipDeviceGetAttribute(&maxGridDimZ, hipDeviceAttributeMaxGridDimZ, deviceID));
 

  dim3 gridSize;
  if (numDimensions == 2) gridSize = {(mesh -> getNx() + blockSize.x - 1)/blockSize.x, (mesh -> getNy() + blockSize.y - 1)/blockSize.y};

  return gridSize;
}

template <typename PRECISION>
__global__ void FLB::CudaUtils::save2DDataToOpenGL(PRECISION* v, hipSurfaceObject_t d_SurfaceTexture)
{
  const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int y = threadIdx.y + blockIdx.y *blockDim.y;
  const unsigned int idx = x + y * FLB::d_Nx;
  
  if (x >= FLB::d_Nx || y >= FLB::d_Ny) return;
  float2 data = make_float2(v[idx], v[idx + FLB::d_N]);
  //printf("OPENGL vx = %6.4f  y = %6.4f\n", v[idx], v[idx + d_N]);
  surf2Dwrite(data, d_SurfaceTexture, x * sizeof(float2), y);
}

template __global__ void FLB::CudaUtils::save2DDataToOpenGL<float>(float *v, hipSurfaceObject_t d_SurfaceTexture);

template __global__ void FLB::CudaUtils::save2DDataToOpenGL<double>(double *v, hipSurfaceObject_t d_SurfaceTexture);

